#include <stdio.h>
#include <stdint.h>

#include "kernel.cu"

int main(int argc, char* argv[])
{
	float *in_h;
	float *in_d;
	unsigned int num_elements, block_size;
	unsigned int mode;

	if (argc == 1) {
		num_elements = 1000;
		block_size = 64;
		mode = 0;
	}
	else if (argc == 2) {
		num_elements = atoi(argv[1]);
		block_size = 64;
		mode = 0;
	}
	else if (argc == 3) {
		num_elements = atoi(argv[1]);
		block_size = atoi(argv[2]);
		mode = 0;
	}
	else if (argc == 4) {
		num_elements = atoi(argv[1]);
		block_size = atoi(argv[2]);
		mode = atoi(argv[3]);
		if (mode != 0 && mode != 1)
		{
			printf("ERROR: Mode can only be an integer within [0, 1]!");
			exit(0);
		}
	}
	else {
		printf("\n    Invalid input parameters!"
			"\n    Usage: ./atmem_bench            # Number of elements: 1,000\tBlock size: 64\tMode: 0"
			"\n    Usage: ./atmem_bench <n>        # Number of elements: n\tBlock size: 64\tMode: 0"
			"\n    Usage: ./atmem_bench <n> <b>    # Number of elements: n\tBlock size: b\tMode: 0"
			"\n    Usage: ./atmem_bench <n> <b> <m>    # Number of elements: n\tMode: m (0=Baseline, 1=Atomic)"
			"\n");
		exit(0);
	}
	// Print all inputs
	printf("Number of elements = %u\nBlock size = %u\nMode = %s (%d)\n", num_elements, block_size, (mode == 0) ? "BASELINE" : "ATOMIC", mode);

	// Host array
	in_h = (float*)malloc(num_elements*sizeof(float));

	printf("Input: ");
	for (int i = 0; i<num_elements; i++)
	{
		in_h[i] = (float)(rand() % 1000) / 100.0;
		printf("%f ", in_h[i]);
	}
	printf("\n");

	printf("Allocating device variables...\n");

	hipMalloc((void**)&in_d, num_elements * sizeof(float));
	hipDeviceSynchronize();

	// H to D
	printf("Copying data from host to device...\n");
	hipMemcpy(in_d, in_h, num_elements * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	// Kernel Launch
	printf("Launching kernel...\n");

	atmem_bench(in_d, num_elements, block_size, mode);

	// D to H
	printf("Copying data from device to host...\n");
	hipMemcpy(in_h, in_d, num_elements * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// Print output
	printf("Output: ");
	for (int i = 0; i<num_elements; i++)
	{
		printf("%f ", in_h[i]);
	}
	printf("\n");

	hipFree(in_d);
	free(in_h);

	return 0;
}
