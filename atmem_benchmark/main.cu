#include <stdio.h>
#include <stdint.h>

#include "kernel.cu"

int main(int argc, char* argv[])
{

    printf("\nSetting up the problem..."); fflush(stdout);

    float *in_h;
    float *in_d;
    unsigned int num_elements;

    if(argc == 1) {
        num_elements = 1000;
    } else if(argc == 2) {
        num_elements = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./atmem_bench            # Input: 1,000"
           "\n    Usage: ./atmem_bench <m>        # Input: m"
           "\n");
        exit(0);
    }

    // Host array
    in_h = (float*) malloc(num_elements*sizeof(float));

    printf("Input: ");
    for(int i=0; i<num_elements;i++)
    {
	in_h[i] = (float)(rand()%1000) / 100.0;
	printf("%f ", in_h[i]);
    }
    printf("\n");
    printf("Array size = %u\n", num_elements);

    printf("Allocating device variables...\n");

    hipMalloc((void**)&in_d, num_elements * sizeof(float));
    hipDeviceSynchronize();

    // H to D
    printf("Copying data from host to device...\n");
    hipMemcpy(in_d, in_h, num_elements * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Kernel Launch
    printf("Launching kernel...\n");

    atmem_bench(in_d, num_elements);

    // D to H
    printf("Copying data from device to host...\n");
    hipMemcpy(in_h, in_d, num_elements * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Print output
    printf("Output: ");
    for(int i=0; i<num_elements;i++)
    {
	printf("%f ", in_h[i]);
    }
    printf("\n");

    hipFree(in_d);
    free(in_h);

    return 0;
}

