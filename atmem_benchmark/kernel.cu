#include "hip/hip_runtime.h"
#define IS_ARRAY_PRINT_ENABLED 0
typedef signed long long TimeType;
/******************************************************************************
* Host Functions
*******************************************************************************/

TimeType find_max(TimeType* time_array, int length)
{
	TimeType result = time_array[0];
	for (int i = 1; i < length; i++)
	{
		if (result < time_array[i]) result = time_array[i];
	}
	return result;
}

/******************************************************************************
* Kernels
*******************************************************************************/
// Mode 0
__global__ void lmwTest_baseline(float* data, float scalar, int blockSize, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	int index = (blockIdx.x*blockDim.x + threadIdx.x)*blockSize;
	for (int i = 0; i<blockSize; i++) {
		data[index + i] += scalar;
	}
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %u\n", temp);
	}
	__threadfence();
}

// Mode 1
__global__ void lmwTest_atomic(float* data, float scalar, int blockSize, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	int index = (blockIdx.x*blockDim.x + threadIdx.x)*blockSize;
	for (int i = 0; i<blockSize; i++) {
		atomicAdd(&(data[index + i]), scalar);
	}
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %lld\n", temp);
	}
	__threadfence();
}

// Mode 2: One thread performs one atomic add on one element
__global__ void oneThreadOneVarAtomicAdd(float* data, float scalar, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	if (threadIdx.x == 0)
	{
		atomicAdd(&(data[0]), scalar);
	}
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %lld\n", temp);
	}
	__threadfence();
}

// Mode 3: All threads in a warp perform one atomic add on one element
__global__ void oneWarpOneVarAtomicAdd(float* data, float scalar, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	atomicAdd(&(data[0]), scalar);
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %lld\n", temp);
	}
	__threadfence();
}

// Mode 4: Every thread in a warp performs one atomic add on one element (i.e. 32 elements in total)
__global__ void oneWarp32VarAtomicAdd(float* data, float scalar, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	atomicAdd(&data[threadIdx.x], scalar);
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %lld\n", temp);
	}
	__threadfence();
}

// Mode 5: Like Mode 4, but the elements are far from one another.
__global__ void oneWarp32VarAtomicAdd_Far(float* data, float scalar, int interval, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	atomicAdd(&data[interval*threadIdx.x], scalar);
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %lld\n", temp);
	}
	__threadfence();
}

// Mode 6: All elements in the vector perform one atomic add to a corresponding element.
__global__ void vectorAtomicAdd(float* data, float scalar, int length, TimeType* elapsed_time) {
	TimeType start_time, end_time, temp;
	if (threadIdx.x == 0) start_time = clock64();
	__syncthreads();
	// Begin	
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < length)
	{
		atomicAdd(&data[index], scalar);
	}
	// End
	__syncthreads();
	if (threadIdx.x == 0)
	{
		end_time = clock64();
		temp = end_time - start_time;
		elapsed_time[blockIdx.x] = temp;
		// printf("Elapsed time: %lld\n", temp);
	}
	__threadfence();
}
/******************************************************************************
* End of Kernel Function Definitions; proceeding to the invocation section
*******************************************************************************/

void atmem_bench(float* input, unsigned int num_elements, unsigned int memory_block_size, unsigned int thread_block_size, int mode = 0) {
	int num_blocks = (num_elements / memory_block_size) / thread_block_size;
	printf("Number of blocks: %d\n", num_blocks);
	// Setting up time parameters
	TimeType* elapsed_time_d;
	TimeType* elapsed_time_h;
	int elapsed_time_size = num_blocks;
	hipError_t cuda_status;

	elapsed_time_h = (TimeType*)malloc(elapsed_time_size * sizeof(TimeType));
	for (int i = 0; i < elapsed_time_size; i++)
		elapsed_time_h[i] = -1;	// Non-zero random value

	hipMalloc((void**)&elapsed_time_d, elapsed_time_size * sizeof(TimeType));
	hipMemset(elapsed_time_d, 0, elapsed_time_size * sizeof(TimeType));
	hipDeviceSynchronize();

	// Events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Invoking Kernel
	hipEventRecord(start);
	if (mode == 0)
		lmwTest_baseline << < num_blocks, thread_block_size >> > (input, 1.0, thread_block_size, elapsed_time_d);
	else if (mode == 1)
		lmwTest_atomic << < num_blocks, thread_block_size >> > (input, 1.0, thread_block_size, elapsed_time_d);
	else if (mode == 2)
		oneThreadOneVarAtomicAdd << < 1, 1 >> > (input, 1.0, elapsed_time_d);
	else if (mode == 3)
		oneWarpOneVarAtomicAdd << < 1, 32 >> > (input, 1.0, elapsed_time_d);
	else if (mode == 4)
		oneWarp32VarAtomicAdd << < 1, 32 >> > (input, 1.0, elapsed_time_d);
	else if (mode == 5)
		oneWarp32VarAtomicAdd_Far << < 1, 32 >> > (input, 1.0, memory_block_size, elapsed_time_d);
	else if (mode == 6)
		vectorAtomicAdd << < num_blocks, thread_block_size >> > (input, 1.0, num_elements, elapsed_time_d);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float total_elapsed_time = 0;
	hipEventElapsedTime(&total_elapsed_time, start, stop);

	hipDeviceSynchronize();

	// Copying time to host
	cuda_status = hipMemcpy(elapsed_time_h, elapsed_time_d, elapsed_time_size * sizeof(TimeType), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess)
	{
		printf("\n*CUDA Error %d during hipMemcpy(D -> H)!*\n\n", cuda_status);
	}
	hipDeviceSynchronize();
	//for (int i = 0; i < num_blocks; i++)
	//	printf("%d\n", elapsed_time_h[i]);

	printf("Total elapsed kernel time %f ms\n", total_elapsed_time);
	printf("Max in-SM cycles: %d cycles\n", find_max(elapsed_time_h, elapsed_time_size));

	free(elapsed_time_h);
	hipFree(elapsed_time_d);
}
